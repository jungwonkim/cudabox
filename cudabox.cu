
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <hiprand/hiprand_kernel.h>

#define RED     "\033[22;31m"
#define GREEN   "\033[22;32m"
#define YELLOW  "\033[22;33m"
#define BLUE    "\033[22;34m"
#define PURPLE  "\033[22;35m"
#define CYAN    "\033[22;36m"
#define GRAY    "\033[22;37m"
#define RESET   "\x1b[m"

#define _cudaerr(cudafn) do { hipError_t err = cudafn; if (err != hipSuccess) { printf("[%d:%s] CUDA_ERROR[%d] %s\n", __LINE__, __func__, err, hipGetErrorString(err)); fflush(stdout); } } while (0)
#define _check()         do { printf(PURPLE"[%d:%s] " RESET "\n", __LINE__, __func__); fflush(stdout); } while (0)
#define _info(fmt,  ...) do { printf(fmt "\n", __VA_ARGS__); fflush(stdout); } while (0)
#define _trace(fmt, ...) do { printf(BLUE  "[%d:%s] " fmt RESET "\n", __LINE__, __func__, __VA_ARGS__); fflush(stdout); } while (0)
#define _debug(fmt, ...) do { printf(GREEN "[%d:%s] " fmt RESET "\n", __LINE__, __func__, __VA_ARGS__); fflush(stdout); } while (0)
#define _error(fmt, ...) do { printf(RED   "[%d:%s] " fmt RESET "\n", __LINE__, __func__, __VA_ARGS__); fflush(stdout); } while (0)

#define RUN_KERNEL_INIT(KERNEL) int kernel_idx = 0; int kernel_found = 0; char* kernel = KERNEL;
#define RUN_KERNEL1(FUNC_NAME, ARG1) \
    if (!kernel_found && strcmp(KERNELS[kernel_idx++], kernel) == 0) { FUNC_NAME<int>   ((int*)    ARG1); kernel_found = 1; } \
    if (!kernel_found && strcmp(KERNELS[kernel_idx++], kernel) == 0) { FUNC_NAME<float> ((float*)  ARG1); kernel_found = 1; } \
    if (!kernel_found && strcmp(KERNELS[kernel_idx++], kernel) == 0) { FUNC_NAME<double>((double*) ARG1); kernel_found = 1; } 
#define RUN_KERNEL2(FUNC_NAME, ARG1, ARG2) \
    if (!kernel_found && strcmp(KERNELS[kernel_idx++], kernel) == 0) { FUNC_NAME<int>   ((int*)    ARG1, (int*)    ARG2); kernel_found = 1; } \
    if (!kernel_found && strcmp(KERNELS[kernel_idx++], kernel) == 0) { FUNC_NAME<float> ((float*)  ARG1, (float*)  ARG2); kernel_found = 1; } \
    if (!kernel_found && strcmp(KERNELS[kernel_idx++], kernel) == 0) { FUNC_NAME<double>((double*) ARG1, (double*) ARG2); kernel_found = 1; }
#define RUN_KERNEL3(FUNC_NAME, ARG1, ARG2, ARG3) \
    if (!kernel_found && strcmp(KERNELS[kernel_idx++], kernel) == 0) { FUNC_NAME<int>   ((int*)    ARG1, (int*)    ARG2, (int*)    ARG3); kernel_found = 1; } \
    if (!kernel_found && strcmp(KERNELS[kernel_idx++], kernel) == 0) { FUNC_NAME<float> ((float*)  ARG1, (float*)  ARG2, (float*)  ARG3); kernel_found = 1; } \
    if (!kernel_found && strcmp(KERNELS[kernel_idx++], kernel) == 0) { FUNC_NAME<double>((double*) ARG1, (double*) ARG2, (double*) ARG3); kernel_found = 1; }
#define RUN_KERNEL4(FUNC_NAME, ARG1, ARG2, ARG3, ARG4) \
    if (!kernel_found && strcmp(KERNELS[kernel_idx++], kernel) == 0) { FUNC_NAME<int>   ((int*)    ARG1, (int*)    ARG2, (int*)    ARG3, ARG4); kernel_found = 1; } \
    if (!kernel_found && strcmp(KERNELS[kernel_idx++], kernel) == 0) { FUNC_NAME<float> ((float*)  ARG1, (float*)  ARG2, (float*)  ARG3, ARG4); kernel_found = 1; } \
    if (!kernel_found && strcmp(KERNELS[kernel_idx++], kernel) == 0) { FUNC_NAME<double>((double*) ARG1, (double*) ARG2, (double*) ARG3, ARG4); kernel_found = 1; }
#define RUN_KERNEL_CHECK() { if (!kernel_found) { _info("%-10s no kernel", kernel); continue; } }

#define MEGA (1024 * 1024UL)

size_t MEMSIZE = 1 * 1024 * MEGA;
int BLOCKSIZE = 256;
int STRIDE = 2;
int SEED = 0;

const char* KERNELS[] = {
  "icomp", "scomp", "dcomp",
  "icudf", "scudf", "dcudf",
  "igevv", "sgevv", "dgevv",
  "istvv", "sstvv", "dstvv",
  "iirvv", "sirvv", "dirvv",
  "iisvv", "sisvv", "disvv",
  "igemv", "sgemv", "dgemv",
  "igemm", "sgemm", "dgemm",
  "irand", "srand", "drand",
};

template <typename T>
__global__ void comp(T* a) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  T sum = x;
  for (int i = 0; i < blockDim.x / 4; i++) {
    sum += i * (i + 13) / (i - 7);
  }
  a[x] = sum;
}

template <typename T>
void run_comp(T* a) {
  int N = MEMSIZE / sizeof(double);
  int B = BLOCKSIZE;
  int G = N / B;
  comp<T><<<G, B>>>(a);
}

template <typename T>
__global__ void cudf(T* a) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  T sum = 0;
  for (int i = 0; i < blockDim.x / 4; i++) {
    sum += i * (i + 10) - (i / 7);
  }
  a[x] = sum;
}

template <typename T>
void run_cudf(T* a) {
  int N = MEMSIZE / sizeof(double);
  int B = BLOCKSIZE;
  int G = N / B;
  cudf<T><<<G, B>>>(a);
}

template <typename T>
__global__ void gevv(T* c, T *a, T *b) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  c[x] = a[x] + b[x];
}

template <typename T>
void run_gevv(T* c, T* a, T* b) {
  int N = MEMSIZE / sizeof(double);
  int B = BLOCKSIZE;
  int G = N / B;
  gevv<T><<<G, B>>>(c, a, b);
}

template <typename T>
__global__ void irvv(T *c, T *a, T *b, int* r) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int i = r[x];
  c[i] = a[i] + b[i];
}

template <typename T>
void run_irvv(T* c, T* a, T* b, int* r) {
  int N = MEMSIZE / sizeof(double);
  int B = BLOCKSIZE;
  int G = N / B;
  irvv<T><<<G, B>>>(c, a, b, r);
}

template <typename T>
__global__ void isvv(T *c, T *a, T *b, int* s) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int i = s[x];
  c[i] = a[i] + b[i];
}

template <typename T>
void run_isvv(T* c, T* a, T* b, int* s) {
  int N = MEMSIZE / sizeof(double);
  int B = BLOCKSIZE;
  int G = N / B;
  isvv<T><<<G, B>>>(c, a, b, s);
}

template <typename T>
__global__ void gemv(T* c, T* a, T* b, int k) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i = 0; i < k; i++) {
    c[x] += a[x * k + i] * b[i];
  }
}

template <typename T>
void run_gemv(T* c, T* a, T* b) {
  int N = sqrt(MEMSIZE);
  int B = BLOCKSIZE;
  int G = N / B;
  gemv<T><<<G, B>>>(c, a, b, N);
}

template <typename T>
__global__ void gemm(T* c, T* a, T* b, int k) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  for(int i = 0; i < k; i++) {
    c[y * k + x] += a[y * k + i] * b[i * k + x];
  }
}

template <typename T>
void run_gemm(T* c, T* a, T* b) {
  int N = sqrt(MEMSIZE / sizeof(double) / sizeof(double));
  dim3 B(sqrt(BLOCKSIZE), sqrt(BLOCKSIZE));
  dim3 G(N / B.x, N / B.y);
  gemm<T><<<G, B>>>(c, a, b, N);
}

template <typename T>
__global__ void rand(T *a, int n) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandState state;
  hiprand_init(0, x, 0, &state);
  int i = hiprand(&state) % n;
  a[i] += i;
}

template <typename T>
void run_rand(T* a) {
  int N = MEMSIZE / sizeof(double) / 8;
  int B = BLOCKSIZE;
  int G = N / B;
  rand<T><<<G, B>>>(a, N);
}

template <typename T>
__global__ void stvv(T *c, T *a, T *b, int stride) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int i = x * stride;
  c[i] = a[i] + b[i];
}

template <typename T>
void run_stvv(T* c, T* a, T* b) {
  int N = MEMSIZE / sizeof(double) / STRIDE;
  int B = BLOCKSIZE;
  int G = N / B;
  stvv<T><<<G, B>>>(c, a, b, STRIDE);
}

double now() {
  static double base_sec = -1;
  struct timeval tv;
  gettimeofday(&tv, NULL);
  if (base_sec < 0) base_sec = tv.tv_sec + 1.e-6 * tv.tv_usec;
  return tv.tv_sec + 1.e-6 * tv.tv_usec - base_sec;
}

int help() {
  printf("Usage: cudabox ");
  for (int i = 0; i < sizeof(KERNELS) / sizeof(char*); i++) printf("%s ", KERNELS[i]);
  printf("\n");
  return 0;
}

int main(int argc, char** argv) {
  if (argc == 2 && (strcmp("help", argv[1]) == 0 || strcmp("-h", argv[1]) == 0)) return help();
  if (getenv("CUDABOX_MEMSIZE"))    MEMSIZE   = atoi(getenv("CUDABOX_MEMSIZE")) * MEGA;
  if (getenv("CUDABOX_BLOCKSIZE"))  BLOCKSIZE = atoi(getenv("CUDABOX_BLOCKSIZE"));
  if (getenv("CUDABOX_STRIDE"))     STRIDE    = atoi(getenv("CUDABOX_STRIDE"));
  if (getenv("CUDABOX_SEED"))       SEED      = atoi(getenv("CUDABOX_SEED"));

  _info("CUDABOX_$ MEMSIZE[%zu]MB BLOCKSIZE[%d] STRIDE[%d] SEED[%d]", MEMSIZE / MEGA, BLOCKSIZE, STRIDE, SEED);

  void *h_a, *h_b, *h_c, *h_a16;
  void *d_a, *d_b, *d_c, *d_a16;

  int *h_r, *h_s;
  int *d_r, *d_s;

  h_a = malloc(MEMSIZE);
  h_b = malloc(MEMSIZE);
  h_c = malloc(MEMSIZE);
  h_r = (int*) malloc(MEMSIZE);
  h_s = (int*) malloc(MEMSIZE);
  h_a16 = malloc(16 * MEMSIZE);

  srand(SEED);
  for (size_t i = 0; i < MEMSIZE / sizeof(int); i++) {
    h_r[i] = rand() % (MEMSIZE / sizeof(double));
    h_s[i] = i;
  }

  _cudaerr(hipFree(0));

  _cudaerr(hipMalloc(&d_a, MEMSIZE));
  _cudaerr(hipMalloc(&d_b, MEMSIZE));
  _cudaerr(hipMalloc(&d_c, MEMSIZE));
  _cudaerr(hipMalloc(&d_c, MEMSIZE));
  _cudaerr(hipMalloc(&d_r, MEMSIZE));
  _cudaerr(hipMalloc(&d_s, MEMSIZE));
  _cudaerr(hipMalloc(&d_a16, 16 * MEMSIZE));

  _cudaerr(hipMemcpy(d_a, h_a, MEMSIZE, hipMemcpyHostToDevice));
  _cudaerr(hipMemcpy(d_b, h_b, MEMSIZE, hipMemcpyHostToDevice));
  _cudaerr(hipMemcpy(d_c, h_c, MEMSIZE, hipMemcpyHostToDevice));
  _cudaerr(hipMemcpy(d_r, h_r, MEMSIZE, hipMemcpyHostToDevice));
  _cudaerr(hipMemcpy(d_s, h_s, MEMSIZE, hipMemcpyHostToDevice));
  _cudaerr(hipMemcpy(d_a16, h_a16, 16 * MEMSIZE, hipMemcpyHostToDevice));

  int all = argc == 1;
  int nkernels = all ? sizeof(KERNELS) / sizeof(char*) : argc - 1;
  char** kernels = all ? (char**) KERNELS : argv + 1;

  for (int i = 0; i < nkernels; i++) {
    double t0 = now();
    RUN_KERNEL_INIT(kernels[i]);
    RUN_KERNEL1(run_comp, d_c);
    RUN_KERNEL1(run_cudf, d_c);
    RUN_KERNEL3(run_gevv, d_c, d_a, d_b);
    RUN_KERNEL3(run_stvv, d_c, d_a, d_b);
    RUN_KERNEL4(run_irvv, d_c, d_a, d_b, d_r);
    RUN_KERNEL4(run_isvv, d_c, d_a, d_b, d_s);
    RUN_KERNEL3(run_gemv, d_c, d_a16, d_b);
    RUN_KERNEL3(run_gemm, d_c, d_a, d_b);
    RUN_KERNEL1(run_rand, d_c);
    RUN_KERNEL_CHECK();
    _cudaerr(hipGetLastError());
    _cudaerr(hipDeviceSynchronize());
    _info("%-10s %lf", kernel, now() - t0);
  }

#ifdef CUDABOX_D2H
  _cudaerr(cudaMemcpy(h_a, d_a, MEMSIZE, cudaMemcpyDeviceToHost));
  _cudaerr(cudaMemcpy(h_b, d_b, MEMSIZE, cudaMemcpyDeviceToHost));
  _cudaerr(cudaMemcpy(h_c, d_c, MEMSIZE, cudaMemcpyDeviceToHost));
  _cudaerr(cudaMemcpy(h_a16, d_a16, 16 * MEMSIZE, cudaMemcpyDeviceToHost));
#endif

  _cudaerr(hipFree(d_a));
  _cudaerr(hipFree(d_b));
  _cudaerr(hipFree(d_c));
  _cudaerr(hipFree(d_r));
  _cudaerr(hipFree(d_s));
  _cudaerr(hipFree(d_a16));

  return 0;
}



#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <sys/time.h>
#include <hiprand/hiprand_kernel.h>

#define __SHORT_FILE__ (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define _cuerror(err) do { if (err != hipSuccess) { printf("[%s:%d:%s] err[%d][%s]\n", __SHORT_FILE__, __LINE__, __func__, err, hipGetErrorString(err)); fflush(stdout); } } while (0)
#define _info(fmt, ...) do { printf(fmt "\n", __VA_ARGS__); fflush(stdout); } while (0)
#define _debug(fmt, ...) do { printf("D [%s:%d:%s] " fmt "\n", __SHORT_FILE__, __LINE__, __func__, __VA_ARGS__); fflush(stdout); } while (0)
#define MEGA (1024 * 1024UL)

hipError_t err;

size_t MEM_SIZE = 1 * 1024 * MEGA;

template <typename T>
__global__ void comp(T* a) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  T sum = 0;
  for (int i = 0; i < blockDim.x / 4; i++) {
    sum += i * (i + 10) - (i / 7);
  }
  a[x] = sum;
}

template <typename T>
void run_comp(T* a) {
  int N = MEM_SIZE / 8;
  int B = 1024;
  int G = N / B;
  comp<T><<<G, B>>>(a);
}

template <typename T>
__global__ void gevv(T* c, T *a, T *b) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  c[x] = a[x] + b[x];
}

template <typename T>
void run_gevv(T* c, T* a, T* b) {
  int N = MEM_SIZE / 8;
  int B = 1024;
  int G = N / B;
  gevv<T><<<G, B>>>(c, a, b);
}

template <typename T>
__global__ void irvv(T *c, T *a, T *b, int* r) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int i = r[x];
  c[i] = a[i] + b[i];
}

template <typename T>
void run_irvv(T* c, T* a, T* b, int* r) {
  int N = MEM_SIZE / 8;
  int B = 1024;
  int G = N / B;
  irvv<T><<<G, B>>>(c, a, b, r);
}

template <typename T>
__global__ void gemv(T* c, T* a, T* b, int k) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i = 0; i < k; i++) {
    c[x] += a[x * k + i] * b[i];
  }
}

template <typename T>
void run_gemv(T* c, T* a, T* b) {
  int N = MEM_SIZE / 1024 / 256;
  int B = 1024;
  int G = N / B;
  gemv<T><<<G, B>>>(c, a, b, N);
}

template <typename T>
__global__ void gemm(T* c, T* a, T* b, int k) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  for(int i = 0; i < k; i++) {
    c[y * k + x] += a[y * k + i] * b[i * k + x];
  }
}

template <typename T>
void run_gemm(T* c, T* a, T* b) {
  int N = MEM_SIZE / 1024 / 256;
  dim3 B(32, 32);
  dim3 G(N / 32, N / 32);
  gemm<T><<<G, B>>>(c, a, b, N);
}

template <typename T>
__global__ void rand(T *a, int n) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandState state;
  hiprand_init(0, x, 0, &state);
  int i = hiprand(&state) % n;
  a[i] += i;
}

template <typename T>
void run_rand(T* a) {
  int N = MEM_SIZE / 8;
  int B = 1024;
  int G = N / B;
  rand<T><<<G, B>>>(a, N);
}

double now() {
  static double base_sec = -1;
  struct timeval tv;
  gettimeofday(&tv, NULL);
  if (base_sec < 0) base_sec = tv.tv_sec + 1.e-6 * tv.tv_usec;
  return tv.tv_sec + 1.e-6 * tv.tv_usec - base_sec;
}

int main(int argc, char** argv) {
  void *h_a, *h_b, *h_c;
  void *d_a, *d_b, *d_c;

  int *h_r;
  int *d_r;

  if (argc > 1) MEM_SIZE = atol(argv[1]) * MEGA;;

  _info("%-10s [%lu]MB [%.2lf]GB", "MEM_SIZE", MEM_SIZE / MEGA, (double) MEM_SIZE / MEGA / 1024);

  h_a = malloc(MEM_SIZE);
  h_b = malloc(MEM_SIZE);
  h_c = malloc(MEM_SIZE);
  h_r = (int*) malloc(MEM_SIZE);

  srand(0);
  for (size_t i = 0; i < MEM_SIZE / sizeof(int); i++) {
    h_r[i] = rand() % (MEM_SIZE / sizeof(double));
  }

  _cuerror(hipFree(0));

  _cuerror(hipMalloc(&d_a, MEM_SIZE));
  _cuerror(hipMalloc(&d_b, MEM_SIZE));
  _cuerror(hipMalloc(&d_c, MEM_SIZE));
  _cuerror(hipMalloc(&d_r, MEM_SIZE));

  _cuerror(hipMemcpy(d_a, h_a, MEM_SIZE, hipMemcpyHostToDevice));
  _cuerror(hipMemcpy(d_b, h_b, MEM_SIZE, hipMemcpyHostToDevice));
  _cuerror(hipMemcpy(d_c, h_c, MEM_SIZE, hipMemcpyHostToDevice));
  _cuerror(hipMemcpy(d_r, h_r, MEM_SIZE, hipMemcpyHostToDevice));

  const char* kernels[] = {
    "icomp", "scomp", "dcomp",
    "igevv", "sgevv", "dgevv",
    "iirvv", "sirvv", "dirvv",
    "igemv", "sgemv", "dgemv",
    "igemm", "sgemm", "dgemm",
    "irand", "srand", "drand",
  };
  int all = argc < 3;
  int nkernels = all ? sizeof(kernels) / sizeof(char*) : argc - 2;

  for (int i = 0; i < nkernels; i++) {
    const char* kernel = all ? kernels[i] : argv[i + 2];
    double t0 = now();
    if      (strcmp(kernels[ 0], kernel) == 0) run_comp<int>   ((int*)    d_c);
    else if (strcmp(kernels[ 1], kernel) == 0) run_comp<float> ((float*)  d_c);
    else if (strcmp(kernels[ 2], kernel) == 0) run_comp<double>((double*) d_c);
    else if (strcmp(kernels[ 3], kernel) == 0) run_gevv<int>   ((int*)    d_c, (int*)    d_a, (int*)    d_b);
    else if (strcmp(kernels[ 4], kernel) == 0) run_gevv<float> ((float*)  d_c, (float*)  d_a, (float*)  d_b);
    else if (strcmp(kernels[ 5], kernel) == 0) run_gevv<double>((double*) d_c, (double*) d_a, (double*) d_b);
    else if (strcmp(kernels[ 6], kernel) == 0) run_irvv<int>   ((int*)    d_c, (int*)    d_a, (int*)    d_b, d_r);
    else if (strcmp(kernels[ 7], kernel) == 0) run_irvv<float> ((float*)  d_c, (float*)  d_a, (float*)  d_b, d_r);
    else if (strcmp(kernels[ 8], kernel) == 0) run_irvv<double>((double*) d_c, (double*) d_a, (double*) d_b, d_r);
    else if (strcmp(kernels[ 9], kernel) == 0) run_gemv<int>   ((int*)    d_c, (int*)    d_a, (int*)    d_b);
    else if (strcmp(kernels[10], kernel) == 0) run_gemv<float> ((float*)  d_c, (float*)  d_a, (float*)  d_b);
    else if (strcmp(kernels[11], kernel) == 0) run_gemv<double>((double*) d_c, (double*) d_a, (double*) d_b);
    else if (strcmp(kernels[12], kernel) == 0) run_gemm<int>   ((int*)    d_c, (int*)    d_a, (int*)    d_b);
    else if (strcmp(kernels[13], kernel) == 0) run_gemm<float> ((float*)  d_c, (float*)  d_a, (float*)  d_b);
    else if (strcmp(kernels[14], kernel) == 0) run_gemm<double>((double*) d_c, (double*) d_a, (double*) d_b);
    else if (strcmp(kernels[15], kernel) == 0) run_rand<int>   ((int*)    d_c);
    else if (strcmp(kernels[16], kernel) == 0) run_rand<float> ((float*)  d_c);
    else if (strcmp(kernels[17], kernel) == 0) run_rand<double>((double*) d_c);
    else { _info("%-10s no kernel", kernel); continue; }
    _cuerror(hipGetLastError());
    _cuerror(hipDeviceSynchronize());
    _info("%-10s %lf", kernel, now() - t0);
  }

  _cuerror(hipMemcpy(h_a, d_a, MEM_SIZE, hipMemcpyDeviceToHost));
  _cuerror(hipMemcpy(h_b, d_b, MEM_SIZE, hipMemcpyDeviceToHost));
  _cuerror(hipMemcpy(h_c, d_c, MEM_SIZE, hipMemcpyDeviceToHost));
  _cuerror(hipMemcpy(h_r, d_r, MEM_SIZE, hipMemcpyDeviceToHost));

  _cuerror(hipFree(d_a));
  _cuerror(hipFree(d_b));
  _cuerror(hipFree(d_c));
  _cuerror(hipFree(d_r));

  return 0;
}

